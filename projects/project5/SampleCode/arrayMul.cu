#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK		128	// number of threads in each block
#endif

#ifndef DATASET_SIZE
#define DATASET_SIZE			( 8*1024*1024 )	// size of the array
#endif

float hA[ DATASET_SIZE ];
float hB[ DATASET_SIZE ];
float hC[ DATASET_SIZE ];

#ifndef TOL
#define TOL		0.00001f		// tolerance to relative error
#endif

void
CudaCheckError( )
{
	hipError_t e = hipGetLastError( );
	if( e != hipSuccess )
	{
		fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
	} 
}

// array multiplication on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C )
{
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	if( gid < DATASET_SIZE )
		C[gid] = A[gid] * B[gid];
}



// main program:

int
main( int argc, char* argv[ ] )
{
	//int dev = findCudaDevice(argc, (const char **)argv);

	// fill host memory:

	for( int i = 0; i < DATASET_SIZE; i++ )
	{
		hA[i] = hB[i] = sqrtf(  (float)i  );
	}

	// allocate device memory:

	float *dA, *dB, *dC;

	hipMalloc( (void **)(&dA), sizeof(hA) );
	hipMalloc( (void **)(&dB), sizeof(hB) );
	hipMalloc( (void **)(&dC), sizeof(hC) );
	CudaCheckError( );

	// copy host memory to the device:

	hipMemcpy( dA, hA, DATASET_SIZE*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dB, hB, DATASET_SIZE*sizeof(float), hipMemcpyHostToDevice );
	CudaCheckError( );

	// setup the execution parameters:

	dim3 grid( DATASET_SIZE / THREADS_PER_BLOCK, 1, 1 );
	dim3 threads( THREADS_PER_BLOCK, 1, 1 );

	// create and start the timer:

	hipDeviceSynchronize( );

	// allocate the events that we'll use for timing:

	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	CudaCheckError( );

	// record the start event:

	hipEventRecord( start, NULL );
	CudaCheckError( );

	// execute the kernel:

	ArrayMul<<< grid, threads >>>( dA, dB, dC );

	// record the stop event:

	hipEventRecord( stop, NULL );
	CudaCheckError( );

	// wait for the stop event to complete:

	hipEventSynchronize( stop );
	CudaCheckError( );

	float msecTotal;
	hipEventElapsedTime( &msecTotal, start, stop );
	CudaCheckError( );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (double)DATASET_SIZE / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "%12d, %4d, %10.2lf\n", DATASET_SIZE, THREADS_PER_BLOCK, megaMultsPerSecond );

	// copy result from the device to the host:

	hipMemcpy( hC, dC, sizeof(hC), hipMemcpyDeviceToHost );
	CudaCheckError( );

	// clean up:

	hipFree( dA );
	hipFree( dB );
	hipFree( dC );
	CudaCheckError( );

	return 0;
}