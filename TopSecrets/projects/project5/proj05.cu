#include "hip/hip_runtime.h"
/*
/usr/local/apps/cuda/cuda-10.1/bin/nvcc -o proj05 proj05.cu ./proj05
*/
// Monte Carlo simulation of a snowball fight:

// system includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	50000
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE	64     // number of threads per block
#endif

#define NUMBLOCKS	( NUMTRIALS / BLOCKSIZE )



// ranges for the random numbers:
// #define PROJECT1

#ifdef PROJECT1
const float TXMIN =	-10.0;	// truck starting location in feet
const float TXMAX =	 10.0;	// truck starting location in feet
const float TYMIN  =	 45.0;	// depth distance to truck in feet
const float TYMAX  =	 55.0;	// depth distance to truck in feet
const float TXVMIN =	 10.0;	// truck x velocity in feet/sec
const float TXVMAX =	 30.0;	// truck x velocity in feet/sec
const float SVMIN  =	 10.0;	// snowball velocity in feet/sec
const float SVMAX  =	 30.0;	// snowball velocity in feet/sec
const float STHMIN = 	 10.0;	// snowball launch angle in degrees
const float STHMAX =	 90.0;	// snowball launch angle in degrees
const float HALFLENMIN = 20.;	// half length of the truck in feet
const float HALFLENMAX = 20.;	// half length of the truck in feet
#else
const float TXMIN =	-10.0;	// truck starting location in feet
const float TXMAX =	 10.0;	// truck starting location in feet
const float TXVMIN =	 15.0;	// truck x velocity in feet/sec
const float TXVMAX =	 35.0;	// truck x velocity in feet/sec
const float TYMIN  =	 40.0;	// depth distance to truck in feet
const float TYMAX  =	 50.0;	// depth distance to truck in feet
const float SVMIN  =	  5.0;	// snowball velocity in feet/sec
const float SVMAX  =	 30.0;	// snowball velocity in feet/sec
const float STHMIN = 	 10.0;	// snowball launch angle in degrees
const float STHMAX =	 70.0;	// snowball launch angle in degrees
const float HALFLENMIN = 15.;	// half length of the truck in feet
const float HALFLENMAX = 30.;	// half length of the truck in feet
#endif


// these are here just to be pretty labels, other than that, they do nothing:
#define IN
#define OUT



// function prototypes:
float		Ranf( float, float );
int		Ranf( int, int );
void		TimeOfDaySeed( );



void
CudaCheckError()
{
	hipError_t e = hipGetLastError();
	if(e != hipSuccess)
	{
   		fprintf( stderr, "Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
	}
}


// degrees-to-radians:
__device__
float
Radians( float d )
{
	return (M_PI/180.f) * d;
}


__global__
void
MonteCarlo( IN float *dtxs, IN float *dtys, IN float *dtxvs, IN float *dsvs, IN float *dsths, IN float *dhalflens, OUT int *dhits )
{
	//unsigned int numItems = blockDim.x;
	//unsigned int wgNum    = blockIdx.x;
	//unsigned int tnum     = threadIdx.x;
	unsigned int gid      = blockIdx.x*blockDim.x + threadIdx.x;

	dhits[gid] = 0;

	// randomize everything:
	float tx	= dtxs[gid];
	float ty	= dtys[gid];
	float txv 	= dtxvs[gid];
	float sv 	= dsvs[gid];
	float sth 	= dsths[gid];
	float sthr	= Radians(sth);
	float svx 	= sv * cos(sthr);
	float svy 	= sv * sin(sthr);

	// how long until the snowball reaches the y depth:
	float t = ty / svy;

	float truckx = tx + txv * t;
	float sbx = svx * t;

	if( fabs(truckx - sbx) < dhalflens[gid] )
	{
		dhits[gid] = 1;
	}
}


// main program:

int
main( int argc, char* argv[ ] )
{
	TimeOfDaySeed( );

	int dev = findCudaDevice(argc, (const char **)argv);

	
	float *htxs  = new float [NUMTRIALS];
	float *htys  = new float [NUMTRIALS];
	float *htxvs = new float [NUMTRIALS];
	float *hsvs  = new float [NUMTRIALS];
	float *hsths = new float [NUMTRIALS];
	float *hhalflens = new float [NUMTRIALS];

	// fill the random-value arrays:
	for( int n = 0; n < NUMTRIALS; n++ )
	{
		htxs[n]  = Ranf(  TXMIN,  TXMAX );
		htys[n]  = Ranf(  TYMIN,  TYMAX );
 		htxvs[n] = Ranf(  TXVMIN, TXVMAX );
 		hsvs[n]  = Ranf(  SVMIN,  SVMAX );
 		hsths[n] = Ranf(  STHMIN, STHMAX );
		hhalflens[n] = Ranf( HALFLENMIN, HALFLENMAX );
	}

	int *hhits = new int [NUMTRIALS];

	// allocate device memory:

	float *dtxs, *dtys, *dtxvs, *dsvs, *dsths, *dhalflens;
	int   *dhits;


	//hipError_t status;
	hipMalloc( (void **)(&dtxs),   NUMTRIALS*sizeof(float) );
	hipMalloc( (void **)(&dtys),   NUMTRIALS*sizeof(float) );
	hipMalloc( (void **)(&dtxvs),   NUMTRIALS*sizeof(float) );
	hipMalloc( (void **)(&dsvs),   NUMTRIALS*sizeof(float) );
	hipMalloc( (void **)(&dsths),   NUMTRIALS*sizeof(float) );
	hipMalloc( (void **)(&dhalflens),   NUMTRIALS*sizeof(float) );
	hipMalloc( (void **)(&dhits),   NUMTRIALS*sizeof(int) );
	CudaCheckError( );


	// copy host memory to the device:
	hipMemcpy( dtxs,  htxs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dtys,  htys,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dtxvs,  htxvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dsvs,  hsvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dsths,  hsths,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dhalflens,  hhalflens,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	CudaCheckError( );


	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid(NUMBLOCKS, 1, 1 );

	// create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	hipEventCreate( &start );
	CudaCheckError( );
	hipEventCreate( &stop );
	CudaCheckError( );

	// record the start event:

	hipEventRecord( start, NULL );
	CudaCheckError( );

	// execute the kernel:

	MonteCarlo<<< grid, threads >>>( dtxs, dtys, dtxvs, dsvs, dsths,  dhalflens, dhits );

	// record the stop event:

	hipEventRecord( stop, NULL );
	CudaCheckError( );

	// wait for the stop event to complete:

	hipEventSynchronize( stop );
	CudaCheckError( );

	float msecTotal = 0.0f;
	hipEventElapsedTime( &msecTotal, start, stop );
	CudaCheckError( );

	// copy result from the device to the host:

	hipMemcpy( hhits, dhits, NUMTRIALS *sizeof(int), hipMemcpyDeviceToHost );
	CudaCheckError( );

	// compute the sum :

	int numHits = 0;
	for (int i = 0; i < NUMTRIALS; i++) {
		numHits += hhits[i];
	}

	float probability = 100.f * (float)numHits / (float)NUMTRIALS;

	// compute and printL

	double secondsTotal = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;
	fprintf( stderr, "Number of Trials, %d, Blocksize, %d, MegaTrials/Second, %.4lf, Probability, %.2f%%\n",
		NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond, probability );

	// clean up memory:
	delete [ ] htxs;
	delete [ ] htys;
	delete [ ] htxvs;
	delete [ ] hsvs;
	delete [ ] hsths;
	delete [ ] hhits;

	hipFree( dtxs );
	CudaCheckError( );
	hipFree( dtys );
	CudaCheckError( );
	hipFree( dtxvs );
	CudaCheckError( );
	hipFree( dsvs );
	CudaCheckError( );
	hipFree( dsths );
	CudaCheckError( );
	hipFree( dhits );
	CudaCheckError( );


	return 0;
}

float
Ranf( float low, float high )
{
	float r = (float) rand();               // 0 - RAND_MAX
	float t = r  /  (float) RAND_MAX;       // 0. - 1.

	return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
	float low = (float)ilow;
	float high = ceil( (float)ihigh );

	return (int) Ranf(low,high);
}

void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}
